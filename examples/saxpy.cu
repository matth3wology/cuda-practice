
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h"

#define N 100000

int main() {
    // Alpha Scalar
    float al = 2.0;

    // X Vector
    float* d_x;
    hipMallocManaged(&d_x, N * sizeof(float));
    for(int i=0;i<N;i++)
        d_x[i] = (float)i;

    // Y Vector
    float* d_y;
    hipMallocManaged(&d_y, N * sizeof(float));
    for(int i=0;i<N;i++)
        d_y[i] = -15.0;
        
    //Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // y = alpha * x + y
    hipblasSaxpy(handle, N, &al, d_x, 1, d_y, 1);
    hipDeviceSynchronize();

    // Print y
    printf("Y: ");
    for(int i=0;i<N;i++)
        printf(" %0.4f ", d_y[i]);
    printf("\n");

    // Cleanup
    hipFree(d_x);
    hipblasDestroy(handle);

    return 0;
}