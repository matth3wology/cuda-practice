
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h"

#define n 1000000
int main(void) {
    
    // Create a Host vector
    float* x;
    x = (float*)malloc(n * sizeof(*x));

    for(int j=0;j<n;j++)
        x[j] = (float)j;

    // Create a Device vector
    float* d_x;
    hipMalloc((void**)&d_x,n*sizeof(*x));

    // Create a cuBLAS
    hipblasHandle_t handle;

    hipblasCreate(&handle);
    hipblasSetVector(n,sizeof(*x),x,1,d_x,1);

    int result;
    hipblasIsamax(handle, n, d_x, 1, &result);
    printf("Max: %d \n", result - 1);

    hipblasIsamin(handle, n, d_x, 1, &result);
    printf("Min: %d \n", result);

    // Clean up the program
    hipFree(d_x);
    hipblasDestroy(handle);
    free(x);
    
    return 0;
}