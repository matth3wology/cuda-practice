#include <stdio.h>
#include "hipblas.h"
#include "hip/hip_runtime.h"

#define IDX2C(i,j,ld) (((j)*(ld))+(i))

#define m 5
#define n 4

int main() {

    // A matrix
    float* a;
    hipMallocManaged(&a, m * n * sizeof(float));
    int ind=11;
    for(int j=0;j<n;j++)
        for(int i=0;i<m;i++)
            a[IDX2C(i,j,m)] = (float)ind++;

    // X Vector
    float* x;
    hipMallocManaged(&x, n * sizeof(float));
    for(int i=0;i<n;i++) x[i] = 1.0f;

    // Y Vector
    float* y;
    hipMallocManaged(&y, m * sizeof(float));
    for(int j=0;j<m;j++) y[j] = 0.0f;

    // Handle cuBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float alpha = 1.0f;
    float beta = 0.0f;

    // y = alpha*a*x + b*y
    hipblasSgemv(handle, HIPBLAS_OP_N, m, n, &alpha, a, m, x, 1, &beta, y, 1);
    hipDeviceSynchronize();

    printf("Y: ");
    for(int i=0;i<m;i++)
        printf(" %0.5f ", y[i]);
    printf("\n");

    // Clean up the program
    hipFree(a);
    hipFree(x);
    hipFree(y);
    hipblasDestroy(handle);
    return 0;
}