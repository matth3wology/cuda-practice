
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h"

#define N 100

int main() {

    // X Vector
    float* d_x;
    hipMallocManaged(&d_x, N * sizeof(float));
    for(int i=0;i<N;i++)
        d_x[i] = (float)i;

    // Y Vector
    float* d_y;
    hipMallocManaged(&d_y, N * sizeof(float));
    for(int i=0;i<N;i++)
        d_y[i] = (float)i;

    //Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float result;
    // y = dot(x,y)
    hipblasSdot(handle, N, d_x, 1, d_y, 1, &result);
    hipDeviceSynchronize();

    // Print y
    printf("Dot Product: %f \n", result);

    // Cleanup
    hipFree(d_x);
    hipFree(d_y);
    hipblasDestroy(handle);

    return 0;
}