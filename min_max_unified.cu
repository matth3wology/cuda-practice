#include <stdio.h>
#include "hipblas.h"

#define n 6
int main(void) {

    
    // Create a Vector and send to the Device
    float* x;
    hipMallocManaged(&x, n*sizeof(*x));
    
    for(int j=0;j<n;j++)
    x[j] = (float)j;
    
    
    // Manage cuBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    int result;
    hipblasIsamax(handle, n, x, 1, &result);
    printf("Max: %d \n", result - 1);

    cublasIsmin(handle, n, x, 1, &result);
    printf("Min: %d \n", result - 1);

    // Clean up the program
    hipFree(x);
    hipblasDestroy(handle);
    
    float result;

    return 0;
}